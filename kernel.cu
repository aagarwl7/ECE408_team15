#include "hip/hip_runtime.h"
#include"support.h"
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>
#include<time.h>

#define norm_vect_len(m) (sqrt(pow((m).x, 2) + pow((m).y, 2))/(m).num_elem)
#define BLOCK_SIZE 1024

__device__ float global_temp[BLOCK_SIZE];
__device__ float global_x[BLOCK_SIZE];
__device__ float global_y[BLOCK_SIZE];

__device__ float calc_delta_nrg(float new_val, float old_val, float *latt, int latt_len) {
	float delta_nrg = 2.*__cosf(new_val-old_val)-2.;
	for(int j = 0; j < latt_len; j++) {
		delta_nrg -= 2*__cosf(new_val-latt[j])-2*__cosf(old_val-latt[j]);
	}
	delta_nrg /= latt_len;
	return delta_nrg;
}
__device__ void perturb_latt(float *latt, int latt_len, int num_steps, float temp, int index, hiprandState s) {
	float old_val = latt[index];
	for(int i = 0; i < num_steps; i++) {
		float new_val = hiprand_uniform(&s) * PI_2;
		float delta_nrg = calc_delta_nrg(new_val, old_val, latt, latt_len);
		if((delta_nrg < 0) || (hiprand_uniform(&s) < exp(-delta_nrg/temp))) {
			latt[index] = new_val;
			old_val = new_val;
		}
	}
}
__global__ void iterate_nrg(int num_temps, float **latt_arr, unsigned int latt_len, int num_steps, hiprandState *states, float *Enrg, float *Magn, int temp_i, float temp) {
	int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  if(index >= latt_len) return;

	hiprandState s;
	if(temp_i == 0) {
		hiprand_init(1234, index, 0, states+index);
		s = states[index];

		for(int i = 0; i < num_temps; i++) {
			latt_arr[i][index] = hiprand_uniform(&s) * PI_2;
		}
	}
	else
		s = states[index];
	
	__syncthreads();
	
	float *latt = latt_arr[temp_i];
	perturb_latt(latt, latt_len, num_steps, temp, index, s);

  __shared__ float local_temp[BLOCK_SIZE/2];
  __shared__ float local_x[BLOCK_SIZE/2];
  __shared__ float local_y[BLOCK_SIZE/2];
  if(threadIdx.x < BLOCK_SIZE/2) {
		local_temp[threadIdx.x] = 0.0;
		local_x[threadIdx.x] = 0.0;
		local_y[threadIdx.x] = 0.0;
	}
	__syncthreads();
 
	// Calculate energy and magnetization
  if(index >= latt_len) return;
	
  float retval = 2.0;
  float spin = latt[index];
	float x=__cosf(spin)/latt_len;
	float y=__sinf(spin)/latt_len;
	
  for(int i = 0; i < latt_len; i++)
    retval -= 2*__cosf(latt[i] - spin);
  retval /= latt_len;
	if(threadIdx.x > BLOCK_SIZE/2) {
		local_temp[threadIdx.x-BLOCK_SIZE/2] = retval;
		local_x[threadIdx.x-BLOCK_SIZE/2] = x;
		local_y[threadIdx.x-BLOCK_SIZE/2] = y;
	}

  __syncthreads();
  for(int stride = 2; stride < BLOCK_SIZE; stride <<= 1) {
    if(threadIdx.x < BLOCK_SIZE/stride) {
      retval += local_temp[threadIdx.x];
			x += local_x[threadIdx.x];
			y += local_y[threadIdx.x];
		}
    __syncthreads();
    if(threadIdx.x >= BLOCK_SIZE/(stride<<1) && threadIdx.x < BLOCK_SIZE/stride) {
      local_temp[threadIdx.x-(BLOCK_SIZE/(stride<<1))] = retval;
      local_x[threadIdx.x-(BLOCK_SIZE/(stride<<1))] = x;
      local_y[threadIdx.x-(BLOCK_SIZE/(stride<<1))] = y;
		}
    __syncthreads();
  }
	if(index == 0) {
		*global_x = 0.;
		*global_y = 0.;
	}
	if(threadIdx.x == 0) {
		atomicAdd(global_x, x);
		atomicAdd(global_y, y);
	}
	__syncthreads();

  if(threadIdx.x == 0) {
		global_temp[blockIdx.x] = retval;
	}
  __syncthreads();
  if(index >= gridDim.x) return;
  for(int stride = 2; stride <= gridDim.x; stride <<= 1) {
    if(index % stride == 0) {
			global_temp[index] += global_temp[index + (stride >> 1)];
		}
		__syncthreads();
  }
	if(index == 0) {
		*Enrg = global_temp[0];
		*Magn = sqrt(pow(global_x[0], 2) + pow(global_y[0], 2));
	}
}

void find_xy_parameters(int num_temps, float **latt_arr, unsigned int latt_len, unsigned int num_steps, float *Enrg, float *Magn) {
  hipError_t cuda_ret;
  dim3 grid_dim = dim3((int)ceil(((float)latt_len)/BLOCK_SIZE), 1, 1);
  dim3 block_dim = dim3(BLOCK_SIZE, 1, 1);

  hiprandState *states;
  cuda_ret = hipMalloc((void **)&states, latt_len*sizeof(hiprandState));
  if(cuda_ret != hipSuccess) FATAL("Unable to alocate device memory");

	for(int i = 0; i < num_temps; i++) {
		float temp = MIN_TEMP+i*TEMP_DIFF/num_temps;
		iterate_nrg<<<grid_dim, block_dim>>>(num_temps, latt_arr, latt_len, num_steps, states, Enrg+i, Magn+i, i, temp);
	}
  hipDeviceSynchronize();
  hipFree(states);

}



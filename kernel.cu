#include "hip/hip_runtime.h"
#include"support.h"
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>
#include<time.h>

#define norm_vect_len(m) (sqrt(pow((m).x, 2) + pow((m).y, 2))/(m).num_elem)
#define BLOCK_SIZE 1024

typedef struct {
  float x;
  float y;
  int num_elem;
} magn_t;


__device__ float global_temp[BLOCK_SIZE];
__global__ void calc_energy(float *latt, unsigned int latt_len, float *nrg) {
  unsigned int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  __shared__ float local_temp[BLOCK_SIZE/2];
  if(threadIdx.x < BLOCK_SIZE/2) local_temp[threadIdx.x] = 0.0;

  if(index >= latt_len) return;
	
  float retval = 2.0;
  float spin = latt[index];
	
  for(int i = 0; i < latt_len; i++)
    retval -= 2*__cosf(latt[i] - spin);
  retval /= latt_len;
	if(threadIdx.x > BLOCK_SIZE/2) local_temp[threadIdx.x-BLOCK_SIZE/2] = retval;
  __syncthreads();
  for(int stride = 2; stride < BLOCK_SIZE; stride <<= 1) {
    if(index < BLOCK_SIZE/stride) 
      retval += local_temp[threadIdx.x];
    __syncthreads();
    if(index >= BLOCK_SIZE/(stride<<1) && index < BLOCK_SIZE/stride)
      local_temp[threadIdx.x-(BLOCK_SIZE/(stride<<1))] = retval;
    __syncthreads();
  }
	
  if(threadIdx.x == 0) {
		global_temp[blockIdx.x] = retval;
	}
  __syncthreads();
	
  if(index >= gridDim.x) return;
  for(int stride = 2; stride < gridDim.x; stride <<= 1) {
    if(index % stride == 0)
      global_temp[index] += global_temp[index + (stride >> 1)];
  }
  __syncthreads();
	
  if(index == 0) {
		*nrg = global_temp[0];
	}

}
__global__ void iterate_nrg(float temp, float *latt, unsigned int latt_len, int num_steps, hiprandState *states, float *nrg) {
  int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  if(index >= latt_len) return;
  //__shared__ float temp_latt[BLOCK_SIZE];
  hiprandState s = states[index];
  hiprand_init(1234, index, 0, &s);

	latt[index] = hiprand_uniform(&s) * PI_2;
	__syncthreads();

  for(int i = 0; i < num_steps; i++) {
    unsigned int rand_ind = index;
    float new_val = hiprand_uniform(&s) * PI_2;
    float old_val = latt[rand_ind];
    float delta_nrg = 2.*__cosf(new_val-latt[rand_ind])-2.;
		/*
    for(int off = 0; off < latt_len; off += BLOCK_SIZE) {
      temp_latt[threadIdx.x] = latt[off+threadIdx.x];
      __syncthreads();
      for(int j = 0; j < BLOCK_SIZE; j++)
        delta_nrg -= 2*__cosf(new_val-temp_latt[j+off])-2*__cosf(old_val-temp_latt[j+off]);
    }
    delta_nrg /= latt_len;
    if(hiprand_uniform(&s) < exp(-delta_nrg/temp) || delta_nrg < 0) {
      latt[rand_ind] = new_val;
    }
		*/
		for(int j = 0; j < latt_len; j++)
			delta_nrg -= 2*__cosf(new_val-latt[j])-2*__cosf(old_val-latt[j]);
		delta_nrg /= latt_len;
    if(hiprand_uniform(&s) < exp(-delta_nrg/temp) || delta_nrg < 0)
      latt[rand_ind] = new_val;
  }
}

void find_xy_parameters(float temp, float *latt, unsigned int latt_len, unsigned int num_steps, float *nrg, float *mag) {
  hipError_t cuda_ret;
  dim3 grid_dim = dim3((int)ceil(((float)latt_len)/BLOCK_SIZE), 1, 1);
  dim3 block_dim = dim3(BLOCK_SIZE, 1, 1);

  hiprandState *states;
  cuda_ret = hipMalloc((void **)&states, latt_len*sizeof(hiprandState));
  if(cuda_ret != hipSuccess) FATAL("Unable to alocate device memory");

  iterate_nrg<<<grid_dim, block_dim>>>(temp, latt, latt_len, num_steps, states, nrg);

  hipDeviceSynchronize();
  calc_energy<<<grid_dim, block_dim>>>(latt, latt_len, nrg);

  hipFree(states);

}



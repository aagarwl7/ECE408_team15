#include "hip/hip_runtime.h"
#include"support.h"

#define norm_vect_len(m) (sqrt(pow((m).x, 2) + pow((m).y, 2))/(m).num_elem)
#define BLOCK_SIZE 1024

typedef struct {
  float x;
  float y;
  int num_elem;
} magn_t;


__device__ float global_temp[BLOCK_SIZE];
__global__ void calc_energy(float *latt, unsigned int latt_len, float *nrg) {
  unsigned int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  __shared__ float local_temp[BLOCK_SIZE/2];
  if(threadIdx.x < BLOCK_SIZE/2) local_temp[threadIdx.x] = 0.0;

  if(index >= latt_len) return;
	
  float retval = 2.0;
  float spin = latt[index];
	
  for(int i = 0; i < latt_len; i++)
    retval -= 2*__cosf(latt[i] - spin);
  retval /= latt_len;
	if(threadIdx.x > BLOCK_SIZE/2) local_temp[threadIdx.x-BLOCK_SIZE/2] = retval;
  __syncthreads();
  for(int stride = 2; stride < BLOCK_SIZE; stride <<= 1) {
    if(index < BLOCK_SIZE/stride) 
      retval += local_temp[threadIdx.x];
    __syncthreads();
    if(index >= BLOCK_SIZE/(stride<<1) && index < BLOCK_SIZE/stride)
      local_temp[threadIdx.x-(BLOCK_SIZE/(stride<<1))] = retval;
    __syncthreads();
  }
	
  if(threadIdx.x == 0) {
		global_temp[blockIdx.x] = retval;
	}
  __syncthreads();

  if(index >= gridDim.x) return;
  for(int stride = 2; stride < gridDim.x; stride <<= 1) {
    if(index % stride == 0)
      global_temp[index] += global_temp[index + (stride >> 1)];
  }
  __syncthreads();

  if(index == 0) {
		*nrg = global_temp[0];
	}

}

__global__ void iterate_nrg(float temp, float *latt, unsigned int latt_len, int *rand_inds, float *rand_elems, float *rands, float *nrg) {
  int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	if(index >= latt_len) return;
  int rand_ind = rand_inds[index];
  float new_val = rand_elems[index];
  float delta_nrg = 2.*__cosf(new_val-latt[rand_ind])-2.;
  for(int i = 0; i < latt_len; i++)
    delta_nrg -= 2*__cosf(new_val-latt[i])-2*__cosf(latt[rand_ind]-latt[i]);
  delta_nrg /= latt_len;
  if(rands[index] < exp(-delta_nrg/temp) || delta_nrg < 0) {
    latt[rand_ind] = new_val;
  }
}
	
void find_xy_parameters(float temp, float *latt, unsigned int latt_len, unsigned int num_steps, float *nrg, float *mag) {
  hipError_t cuda_ret;
  dim3 grid_dim = dim3((int)ceil(((float)latt_len)/BLOCK_SIZE), 1, 1);
  dim3 block_dim = dim3(BLOCK_SIZE, 1, 1);
  
  int arr_len = latt_len;

  int *rand_inds_d;
  float *rand_elems_d, *rands_d;
  cuda_ret = hipMalloc((void **)&rand_inds_d, arr_len * sizeof(int));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
  cuda_ret = hipMalloc((void **)&rand_elems_d, arr_len * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
  cuda_ret = hipMalloc((void **)&rands_d, arr_len * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

  int *rand_ind_arr = (int *)malloc(arr_len*sizeof(float));
  float *rand_spin_arr = (float *)malloc(arr_len*sizeof(float));
  float *rand_arr = (float *)malloc(arr_len*sizeof(float));

  for(int i = 0; i < num_steps; i++) {
      
    for(int j = 0; j < arr_len; j++) {
      rand_ind_arr[j] = j;//rand_latt_ind();
      rand_spin_arr[j] = rand_latt_elem();
      rand_arr[j] = uniform();
    }
    cuda_ret = hipMemcpy(rand_inds_d, rand_ind_arr, arr_len * sizeof(float), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy from host to device");
    cuda_ret = hipMemcpy(rand_elems_d, rand_spin_arr, arr_len * sizeof(float), hipMemcpyHostToDevice);    
    if(cuda_ret != hipSuccess) FATAL("Unable to copy from host to device");
    cuda_ret = hipMemcpy(rands_d, rand_arr, arr_len * sizeof(float), hipMemcpyHostToDevice);    
    if(cuda_ret != hipSuccess) FATAL("Unable to copy from host to device");
    hipDeviceSynchronize();

    iterate_nrg<<<grid_dim, block_dim>>>(temp, latt, latt_len, rand_inds_d, rand_elems_d, rands_d, nrg);
  }
	hipDeviceSynchronize();
  calc_energy<<<grid_dim, block_dim>>>(latt, latt_len, nrg);

	free(rand_ind_arr);
	free(rand_spin_arr);
	free(rand_arr);
}


